#include "hip/hip_runtime.h"
#include <mpi.h>
#include "utils.h"
#include "gpu_expInt.h"

__global__ void calcExpIntegral_mpi(float *res_glob, int n0, int n, int numSamples, int a, float division, int maxIters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ float consts[];

    consts[0] = 0.5772156649015329;
    consts[1] = 1.E-30;
    consts[2] = 3.40282E38;
    consts[3] = maxIters;

    float x = a + (idy+1)*division;

    if(idx < n && idy < numSamples){
        res_glob[idy + idx*numSamples] = calcExp_shared(consts, n0+idx+1, x);
    }
}

extern void GPUexponentialIntegralFloat_mpi(int argc, char **argv, float *results, int block_size_X, int block_size_Y){
    int myid, nprocs, mycard, num_devices;
    int n_loc, s, e;
    float *res_loc, *res_gpu;
    float division;

    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &myid);
    MPI_Comm_size(MPI_COMM_WORLD, &nprocs);

    MPI_Request req[nprocs];
    
    division = (b-a)/numSamples;
    decomp1d(n, nprocs, myid, &s, &e);
    
    n_loc = e-s+1;
 
    num_devices = findBestDevice();

	if (num_devices>1) {
		mycard = myid%num_devices;
		printf("This is process %d, numberOfDevices = %d cardForThisProcess=%d\n", myid, num_devices,mycard);		
		hipSetDevice(mycard);
	}
	
    res_loc = (float *)malloc(n_loc*numSamples*sizeof(float)); 
	hipMalloc((void **) &res_gpu, n_loc*numSamples*sizeof(float));
	
	dim3 dimBlock(block_size_X, block_size_Y);
	dim3 dimGrid((n_loc/dimBlock.x) + (!(n_loc%dimBlock.x)?0:1), 
	                    (numSamples/dimBlock.x) + (!(numSamples%dimBlock.x)?0:1));
    
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    calcExpIntegral_mpi<<<dimGrid,dimBlock,4*sizeof(float)>>>
                    (res_gpu, s, n_loc, numSamples, a, division, maxIters); 
    
    hipMemcpy(res_loc, res_gpu, n_loc*numSamples*sizeof(float), hipMemcpyDeviceToHost);
    
    MPI_Isend(res_loc, numSamples*(e-s+1), MPI_FLOAT, 0, 0, MPI_COMM_WORLD, &req[0]);
    MPI_Wait(&req[0], MPI_STATUS_IGNORE);  
    
    if(myid==0){
        for(int i=0;i<nprocs;i++){
            decomp1d(n, nprocs, i, &s, &e);
            MPI_Irecv(&results[s*numSamples], numSamples*(e-s+1), MPI_FLOAT, i, 0, MPI_COMM_WORLD, &req[i]);
        }
        MPI_Waitall(nprocs, req, MPI_STATUS_IGNORE);
    }

    MPI_Bcast(results, n*numSamples, MPI_FLOAT, 0, MPI_COMM_WORLD);    // couldn't fix make file allow //
                                                              // MPI in C++ files so had to   //
                                                        //send back to all procs to verify results

    hipFree(res_gpu);
    free(res_loc);

    MPI_Finalize();
}
