#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include "utils.h"
#include "gpu_expInt.h"

__device__ float calcExp_simple(int n, float x, int maxIters){
    float eulerConstant=0.5772156649015329;
    float epsilon=1.E-30;
    float bigfloat = 3.40282E38;
    float a,b,c,d,del,fact,h,psi,ans=0.0;
    int i,ii;

    //if( n<0.0 || x<0.0 || (fabsf(x)<epsilon && ((n==0) || (n==1)) ) ) {
    //    std::cout << "Bad arguments were passed to the exponentialIntegral function call" << std::endl;
    //    exit(1);
    //}
    if(n==0){
        ans=expf(-x)/x;
    } else {
        if(x>1.0){
            b=(float)n+x;
            c=bigfloat;
            d=1.0/b;
            h=d;
            for(i=1;i<=maxIters;i++){
                a=(float)(-i)*(n-1+i);
                b+=2.0;
                d=1.0/(a*d+b);
                c=b+a/c;
                del=c*d;
                h*=del;
                if(fabsf(del-1.0)<=epsilon){
                    ans=h*expf(-x);
                    return ans;
                }
            }
            ans=h*expf(-x);
            return ans;
        } else { // Evaluate series
            ans=( (n-1) !=0 ? 1.0/(float)(n-1) : -logf(x)-eulerConstant); // First term
            fact=1.0;
            for(i=1;i<=maxIters;i++){
                fact *= -x/(float)i;
                if(i != (n-1)){
                    del = -fact/(float)(i-n+1);
                } else {
                    psi = -eulerConstant;
                    for(ii=1;ii<=(n-1);ii++){
                        psi += 1.0/(float)ii;
                    }
                    del=fact*(-logf(x)+psi);
                }
                ans+=del;
                if(fabsf(del)<fabsf(ans)*epsilon) return ans;
            }
            return ans;
        }
    }
    return ans;
}

__device__ float calcExp_shared(float *consts, int n, float x){
    float a,b,c,d,del,fact,h,psi,ans=0.0;
    int i,ii;

    //if( n<0.0 || x<0.0 || (fabsf(x)<consts[1] && ((n==0) || (n==1)) ) ) {
    //    std::cout << "Bad arguments were passed to the exponentialIntegral function call" << std::endl;
    //    exit(1);
    //}
    if(n==0){
        ans=expf(-x)/x;
    } else {
        if(x>1.0){
            b=(float)n+x;
            c=consts[2];
            d=1.0/b;
            h=d;
            for(i=1;i<=consts[3];i++){
                a=(float)(-i)*(n-1+i);
                b+=2.0;
                d=1.0/(a*d+b);
                c=b+a/c;
                del=c*d;
                h*=del;
                if(fabsf(del-1.0)<=consts[1]){
                    ans=h*expf(-x);
                    return ans;
                }
            }
            ans=h*expf(-x);
            return ans;
        } else { // Evaluate series
            ans=( (n-1) !=0 ? 1.0/(float)(n-1) : -logf(x)-consts[0]); // First term
            fact=1.0;
            for(i=1;i<=consts[3];i++){
                fact *= -x/(float)i;
                if(i != (n-1)){
                    del = -fact/(float)(i-n+1);
                } else {
                    psi = -consts[0];
                    for(ii=1;ii<=(n-1);ii++){
                        psi += 1.0/(float)ii;
                    }
                    del=fact*(-logf(x)+psi);
                }
                ans+=del;
                if(fabsf(del)<fabsf(ans)*consts[1]) return ans;
            }
            return ans;
        }
    }
    return ans;
}

__global__ void calcExpIntegral_shared(float *res_glob, int n, int numSamples, int a, float division, int maxIters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ float consts[];
    
    consts[0] = 0.5772156649015329;
    consts[1] = 1.E-30;
    consts[2] = 3.40282E38;
    consts[3] = maxIters;

    float x = a + (idy+1)*division;
    
    if(idx<n && idy < numSamples){
        res_glob[idy + idx*numSamples] = calcExp_shared(consts, idx+1, x); 
    }
}

__global__ void calcExpIntegral_glob(float *res_glob, int n, int numSamples, int a, float division, int maxIters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    float x = a + (idy+1)*division;
    
    if(idx<n && idy<numSamples){
        res_glob[idy + idx*numSamples] = calcExp_simple(idx+1, x, maxIters);
    }
}

extern void GPUexponentialIntegralFloat(float *results, int block_size_X, int block_size_Y){
    float *res_glob;
    //size_t results_glob_s;
    //int pitch;
    float division = (b-a)/numSamples;
    
    printf("size of n,numsamples = %d,%d\n",n,numSamples);
    hipMalloc( (void**)&res_glob, n*numSamples*sizeof(float));
    
    dim3 dimBlock(block_size_X, block_size_Y);
    dim3 dimGrid((n/dimBlock.x)+(!(n%dimBlock.x)?0:1),
                (numSamples/dimBlock.y)+(!(numSamples%dimBlock.y)?0:1));
    
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    
    //calcExpIntegral_glob<<<dimGrid, dimBlock>>>(res_glob, n, numSamples, a, division, maxIters);
    calcExpIntegral_shared<<<dimGrid, dimBlock, 4*sizeof(float)>>>(res_glob, n, numSamples, a, division, maxIters);
    
    hipMemcpy(results, res_glob, n*numSamples*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(res_glob);
}
