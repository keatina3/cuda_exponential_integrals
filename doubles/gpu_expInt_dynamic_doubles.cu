#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include "../utils.h"
#include "gpu_expInt_double.h"

__device__ double calcExp_dynamic_double(double *consts, double *dynam_glob, int n, double x){
    double b,c,d,h,ans=0.0;
    double tmp=2.0;
    int i, arr_len=512;
    dim3 dimBlock(arr_len);
    dim3 dimGrid((arr_len/dimBlock.x)+(!(arr_len%dimBlock.x)?0:1));
    
    if(n==0){
        ans=expf(-x)/x;
    } else {
        if(x>1.0){
            b=(double)n+x;
            c=consts[2];
            d=1.0/b;
            h=d; 
            for(i=1; i<consts[3]; i+=arr_len){
                calc_series1_dynamic_double<<<dimGrid,dimBlock,arr_len*sizeof(double)>>>
                                (dynam_glob,i,n,b,c,d);
                tmp = *dynam_glob;
                h *= tmp;
                if(fabs(tmp-1.0)<consts[1])
                    break;
            }
            ans=h*expf(-x);
            return ans;
        } else { // Evaluate series
            ans=( (n-1) !=0 ? 1.0/(double)(n-1) : -logf(x)-consts[0]); // First term
            for(i=1; i<=consts[3]; i+=arr_len){
                calc_series2_dynamic_double<<<dimGrid,dimBlock,arr_len*sizeof(double)>>>
                                (dynam_glob,ans,i,n,x);
                tmp = *dynam_glob;
                ans += tmp;
                if(fabsf(tmp)<consts[1])
                    break;
            }
            return ans;
        }
    }
    return ans;
}

__global__ void calcExpIntegral_dynamic_double(double *res_glob, double *dynam_glob, int n, int numSamples, int a, double division, int maxIters){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    extern __shared__ double consts[];
    
    consts[0] = 0.5772156649015329;
    consts[1] = 1.E-30;
    consts[2] = 3.40282E38;
    consts[3] = maxIters;

    double x = a + (idy+1)*division;

    if(idx<n && idy < numSamples){
        res_glob[idy + idx*numSamples] = calcExp_dynamic_double(consts, 
                                           &dynam_glob[idy + idx*numSamples], n, x);
    }
}

__global__ void calc_series1_dynamic_double(double *h_glob, int iter, int n, double b, double c, double d){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    double a,del,h,epsilon;
    int i,disp;
    extern __shared__ double del_vals[];
    
    disp = iter + idx;
    epsilon = 1.0E-30;

    if(idx<blockDim.x){
        a = (double)(-disp)*(n-1+1);
        b += disp*2.0;
        d = 1.0/(a*d+b);
        c = b+a/c;
        del=c*d;
        del_vals[idx] = del;
        __syncthreads();
        
        if(fabsf(del_vals[0] - 1.0) < epsilon){
            h = 1.0;
            *h_glob = h;
            return;
        }

        i = blockDim.x;     // won't work for non powers of 2 //
        for( ; i>1;i>>=1){
            if(idx<(i/2)){
                del_vals[idx] *= del_vals[idx+(i/2)];
            }
            __syncthreads();
        }
        if(idx==0){
            h = d*del_vals[0];
            *h_glob = h;
        }
    }
}

__global__ void calc_series2_dynamic_double(double *del_sum, double ans, int iter, int n, int x){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int i;
    double fact, epsilon, psi, eulerConstant;
    extern __shared__ double del_vals[];
    
    epsilon = 1.0E-30;
    eulerConstant = 0.5772156649015329;
    
    if(idx<blockDim.x){
        fact = powf(-x/(double)iter, iter);
        if(iter != (n-1)){
            del_vals[idx] = -fact/(double)(iter-n+1);
        } else {
            psi = -eulerConstant;
            for(i=1;i<(n-1);i++){
                psi += 1.0/(double)i;
            }
            del_vals[idx] = fact*(-logf(x)+psi);
        }
        __syncthreads();

        if(fabsf(del_vals[0]<fabsf(ans)*epsilon)){
                *del_sum = 0.0;
                return;
        }
        
        i = blockDim.x;     // won't work for non powers of 2 //
        for( ; i>1;i>>=1){
            if(idx<(i/2)){
                del_vals[idx] += del_vals[idx+(i/2)];
            }
            __syncthreads();
        }
        if(idx==0){
            *del_sum = del_vals[0];
        }
    }
}
